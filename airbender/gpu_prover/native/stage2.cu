#include "hip/hip_runtime.h"
#include "arg_utils.cuh"
#include "ops_complex.cuh"
#include "vectorized.cuh"

using namespace field;
using namespace memory;

using bf = base_field;
using e2 = ext2_field;
using e4 = ext4_field;

// Populates entry-invs and b-cols.
// ENTRY_WIDTH = 1 instantiation computes aggregated entries and multiplicity args for range check 16 lookups.
// ENTRY_WIDTH = 4 instantiation computes aggregated entries and multiplicity args for generic lookups.
template <unsigned ENTRY_WIDTH>
DEVICE_FORCEINLINE void
aggregated_entry_invs_and_multiplicities_arg_kernel(const LookupChallenges *challenges_ptr, matrix_getter<bf, ld_modifier::cs> witness_cols,
                                                    matrix_getter<bf, ld_modifier::cs> setup_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                                    // st_modifier::cg to cache stores for upcoming lookup_a_args_kernel
                                                    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup,
                                                    const unsigned multiplicities_src_cols_start, const unsigned multiplicities_dst_cols_start,
                                                    const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  stage_2_e4_cols.add_col(multiplicities_dst_cols_start);
  witness_cols.add_row(gid);
  witness_cols.add_col(multiplicities_src_cols_start);
  aggregated_entry_invs += gid;

  // for width = 1 (range check) the value is the row index. We can just use gid instead of reading from setup.
  if (ENTRY_WIDTH > 1) {
    setup_cols.add_row(gid);
    setup_cols.add_col(start_col_in_setup);
  }

  const auto [linearization_challenges, gamma] = *challenges_ptr;
  for (unsigned i = 0; i < num_multiplicities_cols; i++) {
    if (i == num_multiplicities_cols - 1 && gid >= num_table_rows_tail) {
      stage_2_e4_cols.set(e4::zero());
      return;
    }

    // for range checks, we can just use gid
    bf val;
    if (ENTRY_WIDTH == 1) {
      val = bf{gid};
    } else {
      val = setup_cols.get();
      setup_cols.add_col(1);
    }
    e4 denom = e4::add(gamma, val);
    if (ENTRY_WIDTH > 1) { // hint to compiler to optimize this part out if possible
#pragma unroll
      for (unsigned j = 1; j < ENTRY_WIDTH; j++) {
        const auto val = setup_cols.get();
        setup_cols.add_col(1);
        denom = e4::add(denom, e4::mul(linearization_challenges[j - 1], val));
      }
    }

    const e4 denom_inv{e4::inv(denom)};

    const auto multiplicity = witness_cols.get();
    stage_2_e4_cols.set(e4::mul(denom_inv, multiplicity));
    aggregated_entry_invs.set(denom_inv);

    witness_cols.add_col(1);
    aggregated_entry_invs += n - 1; // next iteration's warp accesses will be unaligned, but this is likely negligible overall
    stage_2_e4_cols.add_col(1);
  }
}

// TODO (optional): I could unify
// range_check_aggregated_entry_invs_and_multiplicities_arg_kernel
// and
// generic_aggregated_entry_invs_and_multiplicities_arg_kernel
// into a one-size-fits-all kernel by making ENTRY_WIDTH a runtime argument instead of a template parameter.
// I think they're alright as-is.
EXTERN __launch_bounds__(128, 8) __global__ void range_check_aggregated_entry_invs_and_multiplicities_arg_kernel(
    const LookupChallenges *challenges, matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> setup_cols,
    vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
    // st_modifier::cg to cache stores for upcoming lookup_a_args_kernel
    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup, const unsigned multiplicities_src_cols_start,
    const unsigned multiplicities_dst_cols_start, const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {

  aggregated_entry_invs_and_multiplicities_arg_kernel<1>(challenges, witness_cols, setup_cols, stage_2_e4_cols, aggregated_entry_invs, start_col_in_setup,
                                                         multiplicities_src_cols_start, multiplicities_dst_cols_start, num_multiplicities_cols,
                                                         num_table_rows_tail, log_n);
}

EXTERN __launch_bounds__(128, 8) __global__ void generic_aggregated_entry_invs_and_multiplicities_arg_kernel(
    const LookupChallenges *challenges, matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> setup_cols,
    vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
    // st_modifier::cg to cache stores for upcoming lookup_a_args_kernel
    vector_setter<e4, st_modifier::cg> aggregated_entry_invs, const unsigned start_col_in_setup, const unsigned multiplicities_src_cols_start,
    const unsigned multiplicities_dst_cols_start, const unsigned num_multiplicities_cols, const unsigned num_table_rows_tail, const unsigned log_n) {
  aggregated_entry_invs_and_multiplicities_arg_kernel<NUM_LOOKUP_ARGUMENT_KEY_PARTS>(
      challenges, witness_cols, setup_cols, stage_2_e4_cols, aggregated_entry_invs, start_col_in_setup, multiplicities_src_cols_start,
      multiplicities_dst_cols_start, num_multiplicities_cols, num_table_rows_tail, log_n);
}

// This kernel is just for one arg col and should be negligible.
// I'm making it standalone because it doesn't quite fit with the others and
// for easier comparison to zksync_airbender's stage2.rs control flow.
EXTERN __launch_bounds__(128, 8) __global__
    void delegation_aux_poly_kernel(__grid_constant__ const DelegationChallenges challenges, __grid_constant__ const DelegationRequestMetadata request_metadata,
                                    __grid_constant__ const DelegationProcessingMetadata processing_metadata, matrix_getter<bf, ld_modifier::cs> memory_cols,
                                    matrix_getter<bf, ld_modifier::cs> setup_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                    const unsigned delegation_aux_poly_col, const bool handle_delegation_requests, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);
  setup_cols.add_row(gid);

  if (handle_delegation_requests) {
    const bf num = memory_cols.get_at_col(request_metadata.multiplicity_col);

    bf timestamp_low = setup_cols.get_at_col(request_metadata.timestamp_setup_col);
    timestamp_low = bf::add(timestamp_low, request_metadata.in_cycle_write_idx);

    bf timestamp_high = setup_cols.get_at_col(request_metadata.timestamp_setup_col + 1);
    timestamp_high = bf::add(timestamp_high, request_metadata.memory_timestamp_high_from_circuit_idx);

    e4 denom = challenges.gamma;
    denom = e4::add(denom, memory_cols.get_at_col(request_metadata.delegation_type_col));
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[0], memory_cols.get_at_col(request_metadata.abi_mem_offset_high_col)));
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[1], timestamp_low));
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[2], timestamp_high));

    const e4 denom_inv{e4::inv(denom)};
    stage_2_e4_cols.set_at_col(delegation_aux_poly_col, e4::mul(num, denom_inv));
  } else /* process_delegations */ {
    const bf num = memory_cols.get_at_col(processing_metadata.multiplicity_col);

    e4 denom = challenges.gamma;
    denom = e4::add(denom, processing_metadata.delegation_type);
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[0], memory_cols.get_at_col(processing_metadata.abi_mem_offset_high_col)));
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[1], memory_cols.get_at_col(processing_metadata.write_timestamp_col)));
    denom = e4::add(denom, e4::mul(challenges.linearization_challenges[2], memory_cols.get_at_col(processing_metadata.write_timestamp_col + 1)));

    const e4 denom_inv{e4::inv(denom)};
    stage_2_e4_cols.set_at_col(delegation_aux_poly_col, e4::mul(num, denom_inv));
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void lookup_args_kernel(__grid_constant__ const RangeCheckArgsLayout range_check_16_layout,
                            __grid_constant__ const FlattenedLookupExpressionsLayout expressions,
                            __grid_constant__ const FlattenedLookupExpressionsForShuffleRamLayout expressions_for_shuffle_ram,
                            __grid_constant__ const LazyInitTeardownLayout lazy_init_teardown_layout, matrix_getter<bf, ld_modifier::cs> setup_cols,
                            matrix_getter<bf, ld_modifier::cs> witness_cols, matrix_getter<bf, ld_modifier::cs> memory_cols,
                            vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_range_check_16,
                            vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_timestamp_range_checks,
                            vector_getter<e4, ld_modifier::ca> aggregated_entry_invs_for_generic_lookups, const unsigned generic_args_start,
                            const unsigned num_generic_args, matrix_getter<unsigned, ld_modifier::cs> generic_lookups_args_to_table_entries_map,
                            matrix_setter<bf, st_modifier::cs> stage_2_bf_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                            const bf memory_timestamp_high_from_circuit_idx, const unsigned num_stage_2_bf_cols, const unsigned num_stage_2_e4_cols,
                            const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);

  // For bf cols, the final row is reserved for c0 = 0 adjustments.
  // Here we take the opportunity to zero the final row for all stage 2 arg cols.
  if (gid == n - 1) {
    for (unsigned i = 0; i < num_stage_2_bf_cols; i++)
      stage_2_bf_cols.set_at_col(i, bf::zero());
    for (unsigned i = 0; i < num_stage_2_e4_cols; i++)
      stage_2_e4_cols.set_at_col(i, e4::zero());
    return;
  }

  setup_cols.add_row(gid);
  witness_cols.add_row(gid);
  memory_cols.add_row(gid);
  generic_lookups_args_to_table_entries_map.add_row(gid);

  // aggregated_entry_invs loads are uncoalesced e4, but should hit in L2 (or L1 if we're lucky)

  // range check 16 args
  for (unsigned i = 0; i < range_check_16_layout.num_dst_cols; i++) {
    const unsigned src = 2 * i + range_check_16_layout.src_cols_start;
    const bf val0 = bf::into_canonical(witness_cols.get_at_col(src));
    const bf val1 = bf::into_canonical(witness_cols.get_at_col(src + 1));
    const auto entry0 = aggregated_entry_invs_for_range_check_16.get(val0.limb);
    const auto entry1 = aggregated_entry_invs_for_range_check_16.get(val1.limb);
    const auto bf_arg = bf::mul(val0, val1);
    const auto e4_arg = e4::add(entry0, entry1);
    stage_2_bf_cols.set_at_col(range_check_16_layout.bf_args_start + i, bf_arg);
    stage_2_e4_cols.set_at_col(range_check_16_layout.e4_args_start + i, e4_arg);
  }

  // Lookup expressions. These don't use setup cols.
  {
    unsigned i{0}, expression_idx{0}, flat_term_idx{0};
    for (; i < expressions.num_range_check_16_expression_pairs; i++) {
      bf a_and_b[2];
      eval_a_and_b<true>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, expressions.range_check_16_constant_terms_are_zero);
      a_and_b[0] = bf::into_canonical(a_and_b[0]);
      a_and_b[1] = bf::into_canonical(a_and_b[1]);
      const e4 entry_a = aggregated_entry_invs_for_range_check_16.get(a_and_b[0].limb);
      const e4 entry_b = aggregated_entry_invs_for_range_check_16.get(a_and_b[1].limb);
      const bf bf_arg = bf::mul(a_and_b[0], a_and_b[1]);
      const e4 e4_arg = e4::add(entry_a, entry_b);
      stage_2_bf_cols.set_at_col(expressions.bf_dst_cols[i], bf_arg);
      stage_2_e4_cols.set_at_col(expressions.e4_dst_cols[i], e4_arg);
    }

    for (; i < expressions.num_range_check_16_expression_pairs + expressions.num_timestamp_expression_pairs; i++) {
      bf a_and_b[2];
      eval_a_and_b<true>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, expressions.timestamp_constant_terms_are_zero);
      a_and_b[0] = bf::into_canonical(a_and_b[0]);
      a_and_b[1] = bf::into_canonical(a_and_b[1]);
      const e4 entry_a = aggregated_entry_invs_for_timestamp_range_checks.get(a_and_b[0].limb);
      const e4 entry_b = aggregated_entry_invs_for_timestamp_range_checks.get(a_and_b[1].limb);
      const bf bf_arg = bf::mul(a_and_b[0], a_and_b[1]);
      const e4 e4_arg = e4::add(entry_a, entry_b);
      stage_2_bf_cols.set_at_col(expressions.bf_dst_cols[i], bf_arg);
      stage_2_e4_cols.set_at_col(expressions.e4_dst_cols[i], e4_arg);
    }
  }

  // Lookup expressions for shuffle ram. Unlike the expressions above, these may use setup cols.
  for (unsigned i = 0, expression_idx = 0, flat_term_idx = 0; i < expressions_for_shuffle_ram.num_expression_pairs; i++) {
    bf a_and_b[2];
    eval_a_and_b<true>(a_and_b, expressions_for_shuffle_ram, expression_idx, flat_term_idx, setup_cols, witness_cols, memory_cols);
    a_and_b[1] = bf::sub(a_and_b[1], memory_timestamp_high_from_circuit_idx);
    a_and_b[0] = bf::into_canonical(a_and_b[0]);
    a_and_b[1] = bf::into_canonical(a_and_b[1]);
    const e4 entry_a = aggregated_entry_invs_for_timestamp_range_checks.get(a_and_b[0].limb);
    const e4 entry_b = aggregated_entry_invs_for_timestamp_range_checks.get(a_and_b[1].limb);
    const bf bf_arg = bf::mul(a_and_b[0], a_and_b[1]);
    const e4 e4_arg = e4::add(entry_a, entry_b);
    stage_2_bf_cols.set_at_col(expressions_for_shuffle_ram.bf_dst_cols[i], bf_arg);
    stage_2_e4_cols.set_at_col(expressions_for_shuffle_ram.e4_dst_cols[i], e4_arg);
  }

  // 32-bit lazy init address cols, treated as an extra pair of range check 16 cols
  if (lazy_init_teardown_layout.process_shuffle_ram_init) {
    const bf val0 = bf::into_canonical(memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start));
    const bf val1 = bf::into_canonical(memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1));
    const auto entry0 = aggregated_entry_invs_for_range_check_16.get(val0.limb);
    const auto entry1 = aggregated_entry_invs_for_range_check_16.get(val1.limb);
    const auto bf_arg = bf::mul(val0, val1);
    const auto e4_arg = e4::add(entry0, entry1);
    stage_2_bf_cols.set_at_col(lazy_init_teardown_layout.bf_arg_col, bf_arg);
    stage_2_e4_cols.set_at_col(lazy_init_teardown_layout.e4_arg_col, e4_arg);
  }

  // width-3 generic args with fixed table ids. The map makes this simple.
  for (unsigned i = 0; i < num_generic_args; i++) {
    const unsigned absolute_row_index = generic_lookups_args_to_table_entries_map.get_at_col(i);
    const e4 aggregated_entry_inv = aggregated_entry_invs_for_generic_lookups.get(absolute_row_index);
    stage_2_e4_cols.set_at_col(generic_args_start + i, aggregated_entry_inv);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void shuffle_ram_memory_args_kernel(__grid_constant__ const MemoryChallenges challenges, __grid_constant__ const ShuffleRamAccesses shuffle_ram_accesses,
                                        matrix_getter<bf, ld_modifier::cs> setup_cols, matrix_getter<bf, ld_modifier::cs> memory_cols,
                                        vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                        __grid_constant__ const LazyInitTeardownLayout lazy_init_teardown_layout,
                                        const bf memory_timestamp_high_from_circuit_idx, const unsigned memory_args_start, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  setup_cols.add_row(gid);
  memory_cols.add_row(gid);

  // TODO:There's a fair number of e4 x e4 muls and e4 invs here.
  // In theory muls of composite terms could be manually unrolled,
  // helped by precomputed cross-term challenge combinations.
  // It's hard to say what level of unrolling would be optimal.

  // Shuffle ram init
  e4 numerator{challenges.gamma};
  const bf address_low = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start);
  numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));
  const bf address_high = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1);
  numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));

  e4 denom{numerator};
  const bf value_low = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_value_start);
  denom = e4::add(denom, e4::mul(challenges.value_low_challenge, value_low));
  const bf value_high = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_value_start + 1);
  denom = e4::add(denom, e4::mul(challenges.value_high_challenge, value_high));
  const bf timestamp_low = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_timestamp_start);
  denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, timestamp_low));
  const bf timestamp_high = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_timestamp_start + 1);
  denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, timestamp_high));

  // flush result
  e4 num_over_denom_acc = numerator;
  e4 denom_inv{e4::inv(denom)};
  num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
  stage_2_e4_cols.set_at_col(memory_args_start, num_over_denom_acc);

  // Shuffle ram accesses
  // first, read a couple values common across accesses:
  const bf write_timestamp_in_setup_low = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start);
  const bf write_timestamp_in_setup_high = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start + 1);
#pragma unroll 1
  for (unsigned i = 0; i < shuffle_ram_accesses.num_accesses; i++) {
    const auto &access = shuffle_ram_accesses.accesses[i];

    e4 numerator{challenges.gamma};
    const bf address_low = memory_cols.get_at_col(access.address_start);
    numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));

    if (access.is_register_only) {
      numerator = e4::add(numerator, bf::one());
    } else {
      const bf address_high = memory_cols.get_at_col(access.address_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));
      numerator = e4::add(numerator, memory_cols.get_at_col(access.maybe_is_register_start));
      // TODO: It's possible address_high is always zero when memory_cols.get_at_col(access.maybe_is_register_start) is 1, which suggests:
      // const bf is_reg = memory_cols.get_at_col(access.maybe_is_register_start);
      // numerator = e4::add(numerator, memory_cols.get_at_col(access.maybe_is_register_start));
      // if (is_reg.limb) {
      //   const bf address_high = memory_cols.get_at_col(access.address_start + 1);
      //   numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));
      // }
    }

    e4 denom{};

    if (access.is_write) {
      denom = numerator;

      const bf read_value_low = memory_cols.get_at_col(access.read_value_start);
      denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
      const bf read_value_high = memory_cols.get_at_col(access.read_value_start + 1);
      denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));

      const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_start);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
      const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
    } else {
      const bf value_low = memory_cols.get_at_col(access.read_value_start);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
      const bf value_high = memory_cols.get_at_col(access.read_value_start + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));

      denom = numerator;
    }

    const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_start);
    denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
    const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_start + 1);
    denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

    const bf access_index{i};
    const bf write_timestamp_low = bf::add(write_timestamp_in_setup_low, access_index);
    numerator = e4::add(numerator, e4::mul(challenges.timestamp_low_challenge, write_timestamp_low));
    const bf write_timestamp_high = bf::add(write_timestamp_in_setup_high, memory_timestamp_high_from_circuit_idx);
    numerator = e4::add(numerator, e4::mul(challenges.timestamp_high_challenge, write_timestamp_high));

    // flush result
    num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
    e4 denom_inv{e4::inv(denom)};
    num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
    stage_2_e4_cols.set_at_col(memory_args_start + 1 + i, num_over_denom_acc);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void batched_ram_memory_args_kernel(__grid_constant__ const MemoryChallenges challenges, __grid_constant__ const BatchedRamAccesses batched_ram_accesses,
                                        matrix_getter<bf, ld_modifier::cs> memory_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                        const unsigned memory_args_start, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  memory_cols.add_row(gid);

  // Batched ram accesses
  // Compute address_high_contribution, which is common across accesses
  const bf address_high = memory_cols.get_at_col(batched_ram_accesses.abi_mem_offset_high_col);
  const e4 address_high_contribution = e4::mul(address_high, challenges.address_high_challenge);
  // Compute write_timestamp_contribution, also common across accesses
  const bf write_timestamp_low = memory_cols.get_at_col(batched_ram_accesses.write_timestamp_col);
  const e4 write_timestamp_low_contribution = e4::mul(write_timestamp_low, challenges.timestamp_low_challenge);
  const bf write_timestamp_high = memory_cols.get_at_col(batched_ram_accesses.write_timestamp_col + 1);
  const e4 write_timestamp_high_contribution = e4::mul(write_timestamp_high, challenges.timestamp_high_challenge);
  const e4 write_timestamp_contribution = e4::add(write_timestamp_low_contribution, write_timestamp_high_contribution);

  e4 num_over_denom_acc{};
#pragma unroll 1
  for (unsigned i = 0; i < batched_ram_accesses.num_accesses; i++) {
    const auto &access = batched_ram_accesses.accesses[i];
    e4 numerator = e4::add(access.gamma_plus_address_low_contribution, address_high_contribution);

    e4 denom{};

    if (access.is_write) {
      denom = numerator;

      const bf read_value_low = memory_cols.get_at_col(access.read_value_col);
      denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
      const bf read_value_high = memory_cols.get_at_col(access.read_value_col + 1);
      denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));

      const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_col);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
      const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_col + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
    } else {
      const bf value_low = memory_cols.get_at_col(access.read_value_col);
      numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
      const bf value_high = memory_cols.get_at_col(access.read_value_col + 1);
      numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));

      denom = numerator;
    }

    numerator = e4::add(numerator, write_timestamp_contribution);

    const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_col);
    denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
    const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_col + 1);
    denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

    // flush result
    if (i == 0)
      num_over_denom_acc = numerator;
    else
      num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
    e4 denom_inv{e4::inv(denom)};
    num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
    stage_2_e4_cols.set_at_col(memory_args_start + i, num_over_denom_acc);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void register_and_indirect_memory_args_kernel(__grid_constant__ const MemoryChallenges challenges,
                                                  __grid_constant__ const RegisterAndIndirectAccesses register_and_indirect_accesses,
                                                  matrix_getter<bf, ld_modifier::cs> memory_cols, vectorized_e4_matrix_setter<st_modifier::cs> stage_2_e4_cols,
                                                  const unsigned memory_args_start, const unsigned log_n) {
  const unsigned n = 1u << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  // Zeroing the last row for stage 2 bf and e4 args is handled by lookup_args_kernel.
  if (gid >= n - 1)
    return;

  stage_2_e4_cols.add_row(gid);
  stage_2_e4_cols.add_col(memory_args_start);
  memory_cols.add_row(gid);

  // Compute write_timestamp_contribution, common across accesses
  const bf write_timestamp_low = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col);
  const e4 write_timestamp_low_contribution = e4::mul(write_timestamp_low, challenges.timestamp_low_challenge);
  const bf write_timestamp_high = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col + 1);
  const e4 write_timestamp_high_contribution = e4::mul(write_timestamp_high, challenges.timestamp_high_challenge);
  const e4 write_timestamp_contribution = e4::add(write_timestamp_low_contribution, write_timestamp_high_contribution);

  e4 num_over_denom_acc{};
  unsigned flat_indirect_idx = 0;

#pragma unroll 1
  for (unsigned i = 0; i < register_and_indirect_accesses.num_register_accesses; i++) {
    unsigned base_low;
    unsigned base_high;
    // Register contribution
    {
      const auto &register_access = register_and_indirect_accesses.register_accesses[i];

      // TODO: this initial constant contribution could be precomputed and stashed
      e4 numerator = register_access.gamma_plus_one_plus_address_low_contribution;

      e4 denom{};

      if (register_access.is_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(register_access.read_value_col);
        denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
        base_low = bf::into_canonical(read_value_low).limb;
        const bf read_value_high = memory_cols.get_at_col(register_access.read_value_col + 1);
        denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));
        base_high = bf::into_canonical(read_value_high).limb;

        const bf write_value_low = memory_cols.get_at_col(register_access.maybe_write_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(register_access.maybe_write_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(register_access.read_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
        base_low = bf::into_canonical(value_low).limb;
        const bf value_high = memory_cols.get_at_col(register_access.read_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));
        base_high = bf::into_canonical(value_high).limb;

        denom = numerator;
      }

      numerator = e4::add(numerator, write_timestamp_contribution);

      const bf read_timestamp_low = memory_cols.get_at_col(register_access.read_timestamp_col);
      denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(register_access.read_timestamp_col + 1);
      denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

      if (i == 0)
        num_over_denom_acc = numerator;
      else
        num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
      e4 denom_inv{e4::inv(denom)};
      num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
      stage_2_e4_cols.set(num_over_denom_acc);
      stage_2_e4_cols.add_col(1);
    }

    const unsigned lim = flat_indirect_idx + register_and_indirect_accesses.indirect_accesses_per_register_access[i];
#pragma unroll 1
    for (; flat_indirect_idx < lim; flat_indirect_idx++) {
      const auto &indirect_access = register_and_indirect_accesses.indirect_accesses[flat_indirect_idx];

      const unsigned address = base_low + indirect_access.offset;
      const unsigned of = address >> 16;
      const bf address_low = bf{address & 0x0000ffff};
      const bf address_high = bf{base_high + of};

      e4 numerator{challenges.gamma};
      numerator = e4::add(numerator, e4::mul(challenges.address_low_challenge, address_low));
      numerator = e4::add(numerator, e4::mul(challenges.address_high_challenge, address_high));

      e4 denom{};

      if (indirect_access.is_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(indirect_access.read_value_col);
        denom = e4::add(denom, e4::mul(challenges.value_low_challenge, read_value_low));
        const bf read_value_high = memory_cols.get_at_col(indirect_access.read_value_col + 1);
        denom = e4::add(denom, e4::mul(challenges.value_high_challenge, read_value_high));

        const bf write_value_low = memory_cols.get_at_col(indirect_access.maybe_write_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(indirect_access.maybe_write_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(indirect_access.read_value_col);
        numerator = e4::add(numerator, e4::mul(challenges.value_low_challenge, value_low));
        const bf value_high = memory_cols.get_at_col(indirect_access.read_value_col + 1);
        numerator = e4::add(numerator, e4::mul(challenges.value_high_challenge, value_high));

        denom = numerator;
      }

      numerator = e4::add(numerator, write_timestamp_contribution);

      const bf read_timestamp_low = memory_cols.get_at_col(indirect_access.read_timestamp_col);
      denom = e4::add(denom, e4::mul(challenges.timestamp_low_challenge, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(indirect_access.read_timestamp_col + 1);
      denom = e4::add(denom, e4::mul(challenges.timestamp_high_challenge, read_timestamp_high));

      // flush result
      num_over_denom_acc = e4::mul(num_over_denom_acc, numerator);
      e4 denom_inv{e4::inv(denom)};
      num_over_denom_acc = e4::mul(num_over_denom_acc, denom_inv);
      stage_2_e4_cols.set(num_over_denom_acc);
      stage_2_e4_cols.add_col(1);
    }
  }
}
