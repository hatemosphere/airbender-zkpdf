#include "hip/hip_runtime.h"
#include "field.cuh"
#include "memory.cuh"

using namespace field;
using namespace memory;

template <typename T> struct value_getter {
  using value_type = T;
  T value;

  DEVICE_FORCEINLINE T get(const unsigned, const unsigned) const { return value; }
};

using bf = base_field;
using bf_value_getter = value_getter<bf>;
using bf_getter = wrapping_matrix_getter<bf_matrix_getter<ld_modifier::cs>>;
using bf_setter = wrapping_matrix_setter<bf_matrix_setter<st_modifier::cs>>;

using e2 = ext2_field;
using e2_value_getter = value_getter<e2>;
using e2_getter = wrapping_matrix_getter<e2_matrix_getter<ld_modifier::cs>>;
using e2_setter = wrapping_matrix_setter<e2_matrix_setter<st_modifier::cs>>;

using e4 = ext4_field;
using e4_value_getter = value_getter<e4>;
using e4_getter = wrapping_matrix_getter<e4_matrix_getter<ld_modifier::cs>>;
using e4_setter = wrapping_matrix_setter<e4_matrix_setter<st_modifier::cs>>;

using u32 = uint32_t;
using u32_value_getter = value_getter<u32>;
using u32_getter = wrapping_matrix_getter<matrix_getter<u32, ld_modifier::cs>>;
using u32_setter = wrapping_matrix_setter<matrix_setter<u32, st_modifier::cs>>;

using u64 = uint64_t;
using u64_value_getter = value_getter<u64>;
using u64_getter = wrapping_matrix_getter<matrix_getter<u64, ld_modifier::cs>>;
using u64_setter = wrapping_matrix_setter<matrix_setter<u64, st_modifier::cs>>;

template <class T, class U> using unary_fn = U (*)(T);

template <class T0, class T1, class U> using binary_fn = U (*)(T0, T1);

template <class T0, class T1, class T2, class U> using ternary_fn = U (*)(T0, T1, T2);

template <class T, class U> DEVICE_FORCEINLINE void unary_op(const unary_fn<typename T::value_type, typename U::value_type> func, const T arg, U result) {
  const unsigned row = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= result.rows)
    return;
  const unsigned col = blockIdx.y;
  const typename T::value_type arg_value = arg.get(row, col);
  const typename U::value_type result_value = func(arg_value);
  result.set(row, col, result_value);
}

template <class T0, class T1, class U>
DEVICE_FORCEINLINE void binary_op(const binary_fn<typename T0::value_type, typename T1::value_type, typename U::value_type> func, const T0 arg0, const T1 arg1,
                                  U result) {
  const unsigned row = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= result.rows)
    return;
  const unsigned col = blockIdx.y;
  const typename T0::value_type arg0_value = arg0.get(row, col);
  const typename T1::value_type arg1_value = arg1.get(row, col);
  const typename U::value_type result_value = func(arg0_value, arg1_value);
  result.set(row, col, result_value);
}

template <class T0, class T1, class T2, class U>
DEVICE_FORCEINLINE void ternary_op(const ternary_fn<typename T0::value_type, typename T1::value_type, typename T2::value_type, typename U::value_type> func,
                                   const T0 arg0, const T1 arg1, const T2 arg2, U result) {
  const unsigned row = threadIdx.x + blockIdx.x * blockDim.x;
  if (row >= result.rows)
    return;
  const unsigned col = blockIdx.y;
  const typename T0::value_type arg0_value = arg0.get(row, col);
  const typename T1::value_type arg1_value = arg1.get(row, col);
  const typename T2::value_type arg2_value = arg2.get(row, col);
  const typename U::value_type result_value = func(arg0_value, arg1_value, arg2_value);
  result.set(row, col, result_value);
}

template <class T> DEVICE_FORCEINLINE T return_value(const T x) { return x; }

DEVICE_FORCEINLINE bf add(const bf x, const bf y) { return bf::add(x, y); }
DEVICE_FORCEINLINE e2 add(const bf x, const e2 y) { return e2::add(x, y); }
DEVICE_FORCEINLINE e2 add(const e2 x, const bf y) { return e2::add(x, y); }
DEVICE_FORCEINLINE e2 add(const e2 x, const e2 y) { return e2::add(x, y); }
DEVICE_FORCEINLINE e4 add(const bf x, const e4 y) { return e4::add(x, y); }
DEVICE_FORCEINLINE e4 add(const e2 x, const e4 y) { return e4::add(x, y); }
DEVICE_FORCEINLINE e4 add(const e4 x, const bf y) { return e4::add(x, y); }
DEVICE_FORCEINLINE e4 add(const e4 x, const e2 y) { return e4::add(x, y); }
DEVICE_FORCEINLINE e4 add(const e4 x, const e4 y) { return e4::add(x, y); }
DEVICE_FORCEINLINE bf mul(const bf x, const bf y) { return bf::mul(x, y); }
DEVICE_FORCEINLINE e2 mul(const bf x, const e2 y) { return e2::mul(x, y); }
DEVICE_FORCEINLINE e2 mul(const e2 x, const bf y) { return e2::mul(x, y); }
DEVICE_FORCEINLINE e2 mul(const e2 x, const e2 y) { return e2::mul(x, y); }
DEVICE_FORCEINLINE e4 mul(const bf x, const e4 y) { return e4::mul(x, y); }
DEVICE_FORCEINLINE e4 mul(const e2 x, const e4 y) { return e4::mul(x, y); }
DEVICE_FORCEINLINE e4 mul(const e4 x, const bf y) { return e4::mul(x, y); }
DEVICE_FORCEINLINE e4 mul(const e4 x, const e2 y) { return e4::mul(x, y); }
DEVICE_FORCEINLINE e4 mul(const e4 x, const e4 y) { return e4::mul(x, y); }
DEVICE_FORCEINLINE bf sub(const bf x, const bf y) { return bf::sub(x, y); }
DEVICE_FORCEINLINE e2 sub(const bf x, const e2 y) { return e2::sub(x, y); }
DEVICE_FORCEINLINE e2 sub(const e2 x, const bf y) { return e2::sub(x, y); }
DEVICE_FORCEINLINE e2 sub(const e2 x, const e2 y) { return e2::sub(x, y); }
DEVICE_FORCEINLINE e4 sub(const bf x, const e4 y) { return e4::sub(x, y); }
DEVICE_FORCEINLINE e4 sub(const e2 x, const e4 y) { return e4::sub(x, y); }
DEVICE_FORCEINLINE e4 sub(const e4 x, const bf y) { return e4::sub(x, y); }
DEVICE_FORCEINLINE e4 sub(const e4 x, const e2 y) { return e4::sub(x, y); }
DEVICE_FORCEINLINE e4 sub(const e4 x, const e4 y) { return e4::sub(x, y); }

template <class T0, class T1, class T2, class U> DEVICE_FORCEINLINE U mul_add(const T0 x, const T1 y, const T2 z) { return add(mul(x, y), z); }

template <class T0, class T1, class T2, class U> DEVICE_FORCEINLINE U mul_sub(const T0 x, const T1 y, const T2 z) { return sub(mul(x, y), z); }

#define SET_BY_VAL_KERNEL(arg_t)                                                                                                                               \
  EXTERN __global__ void set_by_val_##arg_t##_kernel(const arg_t##_value_getter arg, arg_t##_setter result) { unary_op(return_value, arg, result); }

SET_BY_VAL_KERNEL(u32)
SET_BY_VAL_KERNEL(u64)
SET_BY_VAL_KERNEL(bf)
SET_BY_VAL_KERNEL(e2)
SET_BY_VAL_KERNEL(e4)

#define SET_BY_REF_KERNEL(arg_t)                                                                                                                               \
  EXTERN __global__ void set_by_ref_##arg_t##_kernel(const arg_t##_getter arg, arg_t##_setter result) { unary_op(return_value, arg, result); }

SET_BY_REF_KERNEL(u32)
SET_BY_REF_KERNEL(u64)
SET_BY_REF_KERNEL(bf)
SET_BY_REF_KERNEL(e2)
SET_BY_REF_KERNEL(e4)

#define UNARY_KERNEL(op, arg_t)                                                                                                                                \
  EXTERN __global__ void op##_##arg_t##_kernel(const arg_t##_getter arg, arg_t##_setter result) { unary_op(arg_t::op, arg, result); }

UNARY_KERNEL(dbl, bf)
UNARY_KERNEL(dbl, e2)
UNARY_KERNEL(dbl, e4)
UNARY_KERNEL(inv, bf)
UNARY_KERNEL(inv, e2)
UNARY_KERNEL(inv, e4)
UNARY_KERNEL(neg, bf)
UNARY_KERNEL(neg, e2)
UNARY_KERNEL(neg, e4)
UNARY_KERNEL(sqr, bf)
UNARY_KERNEL(sqr, e2)
UNARY_KERNEL(sqr, e4)

#define PARAMETRIZED_KERNEL(op, arg_t)                                                                                                                         \
  EXTERN __global__ void op##_##arg_t##_kernel(const arg_t##_getter arg, const u32_value_getter parameter, arg_t##_setter result) {                            \
    binary_op(arg_t::op, arg, parameter, result);                                                                                                              \
  }

PARAMETRIZED_KERNEL(pow, bf)
PARAMETRIZED_KERNEL(pow, e2)
PARAMETRIZED_KERNEL(pow, e4)
PARAMETRIZED_KERNEL(shl, bf)
PARAMETRIZED_KERNEL(shl, e2)
PARAMETRIZED_KERNEL(shl, e4)
PARAMETRIZED_KERNEL(shr, bf)
PARAMETRIZED_KERNEL(shr, e2)
PARAMETRIZED_KERNEL(shr, e4)

#define BINARY_KERNEL(op, arg0_t, arg1_t, result_t)                                                                                                            \
  EXTERN __global__ void op##_##arg0_t##_##arg1_t##_kernel(const arg0_t##_getter arg0, const arg1_t##_getter arg1, result_t##_setter result) {                 \
    binary_op(result_t::op, arg0, arg1, result);                                                                                                               \
  }

BINARY_KERNEL(add, bf, bf, bf)
BINARY_KERNEL(add, bf, e2, e2)
BINARY_KERNEL(add, e2, bf, e2)
BINARY_KERNEL(add, e2, e2, e2)
BINARY_KERNEL(add, bf, e4, e4)
BINARY_KERNEL(add, e2, e4, e4)
BINARY_KERNEL(add, e4, bf, e4)
BINARY_KERNEL(add, e4, e2, e4)
BINARY_KERNEL(add, e4, e4, e4)
BINARY_KERNEL(mul, bf, bf, bf)
BINARY_KERNEL(mul, bf, e2, e2)
BINARY_KERNEL(mul, e2, bf, e2)
BINARY_KERNEL(mul, e2, e2, e2)
BINARY_KERNEL(mul, bf, e4, e4)
BINARY_KERNEL(mul, e2, e4, e4)
BINARY_KERNEL(mul, e4, bf, e4)
BINARY_KERNEL(mul, e4, e2, e4)
BINARY_KERNEL(mul, e4, e4, e4)
BINARY_KERNEL(sub, bf, bf, bf)
BINARY_KERNEL(sub, bf, e2, e2)
BINARY_KERNEL(sub, e2, bf, e2)
BINARY_KERNEL(sub, e2, e2, e2)
BINARY_KERNEL(sub, bf, e4, e4)
BINARY_KERNEL(sub, e2, e4, e4)
BINARY_KERNEL(sub, e4, bf, e4)
BINARY_KERNEL(sub, e4, e2, e4)
BINARY_KERNEL(sub, e4, e4, e4)

#define TERNARY_KERNEL(op, arg0_t, arg1_t, arg2_t, result_t)                                                                                                   \
  EXTERN __global__ void op##_##arg0_t##_##arg1_t##_##arg2_t##_kernel(const arg0_t##_getter arg0, const arg1_t##_getter arg1, const arg2_t##_getter arg2,      \
                                                                      result_t##_setter result) {                                                              \
    ternary_op(op, arg0, arg1, arg2, result);                                                                                                                  \
  }

TERNARY_KERNEL(mul_add, bf, bf, bf, bf)
TERNARY_KERNEL(mul_add, bf, bf, e2, e2)
TERNARY_KERNEL(mul_add, bf, bf, e4, e4)
TERNARY_KERNEL(mul_add, bf, e2, bf, e2)
TERNARY_KERNEL(mul_add, bf, e2, e2, e2)
TERNARY_KERNEL(mul_add, bf, e2, e4, e4)
TERNARY_KERNEL(mul_add, bf, e4, bf, e4)
TERNARY_KERNEL(mul_add, bf, e4, e2, e4)
TERNARY_KERNEL(mul_add, bf, e4, e4, e4)
TERNARY_KERNEL(mul_add, e2, bf, bf, e2)
TERNARY_KERNEL(mul_add, e2, bf, e2, e2)
TERNARY_KERNEL(mul_add, e2, bf, e4, e4)
TERNARY_KERNEL(mul_add, e2, e2, bf, e2)
TERNARY_KERNEL(mul_add, e2, e2, e2, e2)
TERNARY_KERNEL(mul_add, e2, e2, e4, e4)
TERNARY_KERNEL(mul_add, e2, e4, bf, e4)
TERNARY_KERNEL(mul_add, e2, e4, e2, e4)
TERNARY_KERNEL(mul_add, e2, e4, e4, e4)
TERNARY_KERNEL(mul_add, e4, bf, bf, e4)
TERNARY_KERNEL(mul_add, e4, bf, e2, e4)
TERNARY_KERNEL(mul_add, e4, bf, e4, e4)
TERNARY_KERNEL(mul_add, e4, e2, bf, e4)
TERNARY_KERNEL(mul_add, e4, e2, e2, e4)
TERNARY_KERNEL(mul_add, e4, e2, e4, e4)
TERNARY_KERNEL(mul_add, e4, e4, bf, e4)
TERNARY_KERNEL(mul_add, e4, e4, e2, e4)
TERNARY_KERNEL(mul_add, e4, e4, e4, e4)
TERNARY_KERNEL(mul_sub, bf, bf, bf, bf)
TERNARY_KERNEL(mul_sub, bf, bf, e2, e2)
TERNARY_KERNEL(mul_sub, bf, bf, e4, e4)
TERNARY_KERNEL(mul_sub, bf, e2, bf, e2)
TERNARY_KERNEL(mul_sub, bf, e2, e2, e2)
TERNARY_KERNEL(mul_sub, bf, e2, e4, e4)
TERNARY_KERNEL(mul_sub, bf, e4, bf, e4)
TERNARY_KERNEL(mul_sub, bf, e4, e2, e4)
TERNARY_KERNEL(mul_sub, bf, e4, e4, e4)
TERNARY_KERNEL(mul_sub, e2, bf, bf, e2)
TERNARY_KERNEL(mul_sub, e2, bf, e2, e2)
TERNARY_KERNEL(mul_sub, e2, bf, e4, e4)
TERNARY_KERNEL(mul_sub, e2, e2, bf, e2)
TERNARY_KERNEL(mul_sub, e2, e2, e2, e2)
TERNARY_KERNEL(mul_sub, e2, e2, e4, e4)
TERNARY_KERNEL(mul_sub, e2, e4, bf, e4)
TERNARY_KERNEL(mul_sub, e2, e4, e2, e4)
TERNARY_KERNEL(mul_sub, e2, e4, e4, e4)
TERNARY_KERNEL(mul_sub, e4, bf, bf, e4)
TERNARY_KERNEL(mul_sub, e4, bf, e2, e4)
TERNARY_KERNEL(mul_sub, e4, bf, e4, e4)
TERNARY_KERNEL(mul_sub, e4, e2, bf, e4)
TERNARY_KERNEL(mul_sub, e4, e2, e2, e4)
TERNARY_KERNEL(mul_sub, e4, e2, e4, e4)
TERNARY_KERNEL(mul_sub, e4, e4, bf, e4)
TERNARY_KERNEL(mul_sub, e4, e4, e2, e4)
TERNARY_KERNEL(mul_sub, e4, e4, e4, e4)
